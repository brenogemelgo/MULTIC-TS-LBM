#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuPhi(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    float pop[GLINKS];
    pop[0] = d.g[gpu_idx_global4(x,y,z,0)];
    pop[1] = d.g[gpu_idx_global4(x,y,z,1)];
    pop[2] = d.g[gpu_idx_global4(x,y,z,2)];
    pop[3] = d.g[gpu_idx_global4(x,y,z,3)];
    pop[4] = d.g[gpu_idx_global4(x,y,z,4)];
    pop[5] = d.g[gpu_idx_global4(x,y,z,5)];
    pop[6] = d.g[gpu_idx_global4(x,y,z,6)];

    const float phi_val = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6];
        
    d.phi[idx3] = phi_val;
}

__global__ void gpuGradients(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    // TODO: D3Q19 for gradients
    const float gradx = 0.375f * (d.phi[gpu_idx_global3(x+1,y,z)] - d.phi[gpu_idx_global3(x-1,y,z)]);
    const float grady = 0.375f * (d.phi[gpu_idx_global3(x,y+1,z)] - d.phi[gpu_idx_global3(x,y-1,z)]);
    const float gradz = 0.375f * (d.phi[gpu_idx_global3(x,y,z+1)] - d.phi[gpu_idx_global3(x,y,z-1)]);
    
    const float phi_val = d.phi[idx3];
    const float grad2 = gradx*gradx + grady*grady + gradz*gradz;
    const float mag = rsqrtf(grad2 + 1e-9f);
    const float normx_val = gradx * mag;
    const float normy_val = grady * mag;
    const float normz_val = gradz * mag;
    const float ind_val = phi_val * (1.0f - phi_val) * (normx_val*normx_val + normy_val*normy_val + normz_val*normz_val);

    d.normx[idx3] = normx_val;
    d.normy[idx3] = normy_val;
    d.normz[idx3] = normz_val;
    d.ind[idx3] = ind_val;
}

__global__ void gpuForces(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float ind_val = d.ind[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];

    //constexpr ci_t H_CIX[19] = { 0, 1,-1, 0, 0, 0, 0, 1,-1, 1,-1, 0, 0, 1,-1, 1,-1, 0, 0 };
    //constexpr ci_t H_CIY[19] = { 0, 0, 0, 1,-1, 0, 0, 1,-1, 0, 0, 1,-1,-1, 1, 0, 0, 1,-1 };
    //constexpr ci_t H_CIZ[19] = { 0, 0, 0, 0, 0, 1,-1, 0, 0, 1,-1, 1,-1, 0, 0,-1, 1,-1, 1 };

    // TODO: D3Q19 for curvature
    float curvature = 0.0f;
    if (ind_val > 0.2f) {
        curvature = -0.375f * (d.normx[gpu_idx_global3(x+1,y,z)] - d.normx[gpu_idx_global3(x-1,y,z)] +
                               d.normy[gpu_idx_global3(x,y+1,z)] - d.normy[gpu_idx_global3(x,y-1,z)] +
                               d.normz[gpu_idx_global3(x,y,z+1)] - d.normz[gpu_idx_global3(x,y,z-1)]);
    }

    const float coeff_force = SIGMA * curvature;
    d.ffx[idx3] = coeff_force * normx_val * ind_val;
    d.ffy[idx3] = coeff_force * normy_val * ind_val;
    d.ffz[idx3] = coeff_force * normz_val * ind_val;
}

__global__ void gpuEvolvePhaseField(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;
        
    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float phi_val = d.phi[idx3];
    const float ux_val = d.ux[idx3];
    const float uy_val = d.uy[idx3];
    const float uz_val = d.uz[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];

    // rest 
    d.g[gpu_idx_global4(x,y,z,0)] = W_G_0 * phi_val;

    // helpers
    const float phi_norm = W_G_1 * GAMMA * phi_val * (1.0f - phi_val);
    const float mult_phi = W_G_1 * phi_val;
    const float a3 = 3.0f * mult_phi;

    // orthogonal 
    float geq = mult_phi + a3 * ux_val;
    float anti_diff = phi_norm * normx_val;
    d.g[gpu_idx_global4(x+1,y,z,1)] = geq + anti_diff;
    
    geq = mult_phi - a3 * ux_val;
    d.g[gpu_idx_global4(x-1,y,z,2)] = geq - anti_diff;

    geq = mult_phi + a3 * uy_val;
    anti_diff = phi_norm * normy_val;
    d.g[gpu_idx_global4(x,y+1,z,3)] = geq + anti_diff;

    geq = mult_phi- a3 * uy_val;
    d.g[gpu_idx_global4(x,y-1,z,4)] = geq - anti_diff;

    geq = mult_phi + a3 * uz_val;
    anti_diff = phi_norm * normz_val;
    d.g[gpu_idx_global4(x,y,z+1,5)] = geq + anti_diff;

    geq = mult_phi - a3 * uz_val;
    d.g[gpu_idx_global4(x,y,z-1,6)] = geq - anti_diff;
} 
