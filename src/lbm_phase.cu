#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuPhi(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    float pop[GLINKS];
    pop[0] = d.g[gpu_idx_global4(x,y,z,0)];
    pop[1] = d.g[gpu_idx_global4(x,y,z,1)];
    pop[2] = d.g[gpu_idx_global4(x,y,z,2)];
    pop[3] = d.g[gpu_idx_global4(x,y,z,3)];
    pop[4] = d.g[gpu_idx_global4(x,y,z,4)];
    pop[5] = d.g[gpu_idx_global4(x,y,z,5)];
    pop[6] = d.g[gpu_idx_global4(x,y,z,6)];

    const float phi_val = pop[0] + pop[1] + pop[2] + pop[3] + pop[4] + pop[5] + pop[6];
        
    d.phi[idx3] = phi_val;
}

__global__ void gpuGradients(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    float w_sum_grad_x = W_1_TO_6  * d.phi[gpu_idx_global3(x+1,y,z)]   - W_1_TO_6  * d.phi[gpu_idx_global3(x-1,y,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y+1,z)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y-1,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y,z-1)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y-1,z)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y+1,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y,z-1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y,z+1)];

    float w_sum_grad_y = W_1_TO_6  * d.phi[gpu_idx_global3(x,y+1,z)]   - W_1_TO_6  * d.phi[gpu_idx_global3(x,y-1,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y+1,z)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y-1,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x,y+1,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x,y-1,z-1)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y+1,z)] - W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y-1,z)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x,y+1,z-1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x,y-1,z+1)];

    float w_sum_grad_z = W_1_TO_6  * d.phi[gpu_idx_global3(x,y,z+1)]   - W_1_TO_6  * d.phi[gpu_idx_global3(x,y,z-1)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y,z-1)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x,y+1,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x,y-1,z-1)]
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x-1,y,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x+1,y,z-1)] 
                       + W_7_TO_18 * d.phi[gpu_idx_global3(x,y-1,z+1)] - W_7_TO_18 * d.phi[gpu_idx_global3(x,y+1,z-1)];
    #ifdef D3Q27
    w_sum_grad_x += W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z-1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z+1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z-1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z+1)];

    w_sum_grad_y += W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z-1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z+1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z-1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z+1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z-1)];

    w_sum_grad_z += W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y-1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y+1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z-1)]
                  + W_19_TO_26 * d.phi[gpu_idx_global3(x-1,y+1,z+1)] - W_19_TO_26 * d.phi[gpu_idx_global3(x+1,y-1,z-1)];
    #endif // D3Q27
        
    const float grad_phi_x = 3.0f * w_sum_grad_x;
    const float grad_phi_y = 3.0f * w_sum_grad_y;
    const float grad_phi_z = 3.0f * w_sum_grad_z;
    
    const float phi_val = d.phi[idx3];
    const float grad2 = grad_phi_x*grad_phi_x + grad_phi_y*grad_phi_y + grad_phi_z*grad_phi_z;
    const float mag = rsqrtf(grad2 + 1e-9f);
    const float normx_val = grad_phi_x * mag;
    const float normy_val = grad_phi_y * mag;
    const float normz_val = grad_phi_z * mag;
    const float ind_val = phi_val * (1.0f - phi_val) * (normx_val*normx_val + normy_val*normy_val + normz_val*normz_val);

    d.normx[idx3] = normx_val;
    d.normy[idx3] = normy_val;
    d.normz[idx3] = normz_val;
    d.ind[idx3] = ind_val;
}

__global__ void gpuForces(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float ind_val = d.ind[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];

    float curvature = 0.0f;
    if (ind_val > 0.2f) {
        float w_sum_curv = W_1_TO_6  *  d.normx[gpu_idx_global3(x+1,y,z)] 
                         - W_1_TO_6  *  d.normx[gpu_idx_global3(x-1,y,z)]
                         + W_1_TO_6  *  d.normy[gpu_idx_global3(x,y+1,z)] 
                         - W_1_TO_6  *  d.normy[gpu_idx_global3(x,y-1,z)]
                         + W_1_TO_6  *  d.normz[gpu_idx_global3(x,y,z+1)] 
                         - W_1_TO_6  *  d.normz[gpu_idx_global3(x,y,z-1)]
                         + W_7_TO_18 * (d.normx[gpu_idx_global3(x+1,y+1,z)] + d.normy[gpu_idx_global3(x+1,y+1,z)]) 
                         - W_7_TO_18 * (d.normx[gpu_idx_global3(x-1,y-1,z)] + d.normy[gpu_idx_global3(x-1,y-1,z)])
                         + W_7_TO_18 * (d.normx[gpu_idx_global3(x+1,y,z+1)] + d.normz[gpu_idx_global3(x+1,y,z+1)]) 
                         - W_7_TO_18 * (d.normx[gpu_idx_global3(x-1,y,z+1)] + d.normz[gpu_idx_global3(x-1,y,z+1)])
                         + W_7_TO_18 * (d.normy[gpu_idx_global3(x,y+1,z+1)] + d.normz[gpu_idx_global3(x,y+1,z+1)]) 
                         - W_7_TO_18 * (d.normy[gpu_idx_global3(x,y-1,z+1)] + d.normz[gpu_idx_global3(x,y-1,z+1)])
                         + W_7_TO_18 * (d.normx[gpu_idx_global3(x+1,y-1,z)] - d.normy[gpu_idx_global3(x+1,y-1,z)]) 
                         - W_7_TO_18 * (d.normx[gpu_idx_global3(x-1,y+1,z)] - d.normy[gpu_idx_global3(x-1,y+1,z)])
                         + W_7_TO_18 * (d.normx[gpu_idx_global3(x+1,y,z-1)] - d.normz[gpu_idx_global3(x+1,y,z-1)]) 
                         - W_7_TO_18 * (d.normx[gpu_idx_global3(x-1,y,z-1)] - d.normz[gpu_idx_global3(x-1,y,z-1)])
                         + W_7_TO_18 * (d.normy[gpu_idx_global3(x,y+1,z-1)] - d.normz[gpu_idx_global3(x,y+1,z-1)]) 
                         - W_7_TO_18 * (d.normy[gpu_idx_global3(x,y-1,z+1)] - d.normz[gpu_idx_global3(x,y-1,z+1)]);
        #ifdef D3Q27
        w_sum_curv += W_19_TO_26 * (d.normx[gpu_idx_global3(x+1,y+1,z+1)] + d.normy[gpu_idx_global3(x+1,y+1,z+1)] + d.normz[gpu_idx_global3(x+1,y+1,z+1)]) 
                    - W_19_TO_26 * (d.normx[gpu_idx_global3(x-1,y-1,z-1)] + d.normy[gpu_idx_global3(x-1,y-1,z-1)] + d.normz[gpu_idx_global3(x-1,y-1,z-1)])
                    + W_19_TO_26 * (d.normx[gpu_idx_global3(x+1,y+1,z-1)] + d.normy[gpu_idx_global3(x+1,y+1,z-1)] - d.normz[gpu_idx_global3(x+1,y+1,z-1)]) 
                    - W_19_TO_26 * (d.normx[gpu_idx_global3(x-1,y-1,z+1)] + d.normy[gpu_idx_global3(x-1,y-1,z+1)] - d.normz[gpu_idx_global3(x-1,y-1,z+1)])
                    + W_19_TO_26 * (d.normx[gpu_idx_global3(x+1,y-1,z+1)] - d.normy[gpu_idx_global3(x+1,y-1,z+1)] + d.normz[gpu_idx_global3(x+1,y-1,z+1)]) 
                    - W_19_TO_26 * (d.normx[gpu_idx_global3(x-1,y+1,z-1)] - d.normy[gpu_idx_global3(x-1,y+1,z-1)] + d.normz[gpu_idx_global3(x-1,y+1,z-1)])
                    + W_19_TO_26 * (d.normx[gpu_idx_global3(x+1,y-1,z-1)] - d.normy[gpu_idx_global3(x+1,y-1,z-1)] - d.normz[gpu_idx_global3(x+1,y-1,z-1)]) 
                    - W_19_TO_26 * (d.normx[gpu_idx_global3(x-1,y+1,z+1)] - d.normy[gpu_idx_global3(x-1,y+1,z+1)] - d.normz[gpu_idx_global3(x-1,y+1,z+1)]);
        #endif // D3Q27
        curvature = -3.0f * w_sum_curv;   
    }

    const float coeff_force = SIGMA * curvature;
    d.ffx[idx3] = coeff_force * normx_val * ind_val;
    d.ffy[idx3] = coeff_force * normy_val * ind_val;
    d.ffz[idx3] = coeff_force * normz_val * ind_val;
}

__global__ void gpuEvolvePhaseField(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;
        
    const idx_t idx3 = gpu_idx_global3(x,y,z);

    const float phi_val = d.phi[idx3];
    const float ux_val = d.ux[idx3];
    const float uy_val = d.uy[idx3];
    const float uz_val = d.uz[idx3];
    const float normx_val = d.normx[idx3];
    const float normy_val = d.normy[idx3];
    const float normz_val = d.normz[idx3];

    // rest 
    d.g[gpu_idx_global4(x,y,z,0)] = W_G_0 * phi_val;

    // helpers
    const float phi_norm = W_G_1 * GAMMA * phi_val * (1.0f - phi_val);
    const float mult_phi = W_G_1 * phi_val;
    const float a3 = 3.0f * mult_phi;

    // orthogonal 
    float geq = mult_phi + a3 * ux_val;
    float anti_diff = phi_norm * normx_val;
    d.g[gpu_idx_global4(x+1,y,z,1)] = geq + anti_diff;
    
    geq = mult_phi - a3 * ux_val;
    d.g[gpu_idx_global4(x-1,y,z,2)] = geq - anti_diff;

    geq = mult_phi + a3 * uy_val;
    anti_diff = phi_norm * normy_val;
    d.g[gpu_idx_global4(x,y+1,z,3)] = geq + anti_diff;

    geq = mult_phi- a3 * uy_val;
    d.g[gpu_idx_global4(x,y-1,z,4)] = geq - anti_diff;

    geq = mult_phi + a3 * uz_val;
    anti_diff = phi_norm * normz_val;
    d.g[gpu_idx_global4(x,y,z+1,5)] = geq + anti_diff;

    geq = mult_phi - a3 * uz_val;
    d.g[gpu_idx_global4(x,y,z-1,6)] = geq - anti_diff;
} 
