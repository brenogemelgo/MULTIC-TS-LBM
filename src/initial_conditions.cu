#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitFields(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    d.ux[idx3] = 0.0f;
    d.uy[idx3] = 0.0f;
    d.uz[idx3] = 0.0f;
    d.phi[idx3] = 0.0f;
    d.rho[idx3] = 1.0f;
    d.ffx[idx3] = 0.0f * 1e-7f;
    d.ffy[idx3] = 0.0f * 1e-5f;
    d.ffz[idx3] = 0.0f * 1e-5f;
    d.normx[idx3] = 0.0f;
    d.normy[idx3] = 0.0f;
    d.normz[idx3] = 0.0f;
    d.pxx[idx3] = 0.0f;
    d.pyy[idx3] = 0.0f;
    d.pzz[idx3] = 0.0f;
    d.pxy[idx3] = 0.0f;
    d.pxz[idx3] = 0.0f;
    d.pyz[idx3] = 0.0f;
}

__global__ void gpuInitJetShape(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = 0;

    if (x >= NX || y >= NY) return;

    const float center_x = (NX-1) * 0.5f;
    const float center_y = (NY-1) * 0.5f;

    const float dx = x-center_x, dy = y-center_y;
    const float radial_dist = sqrtf(dx*dx + dy*dy);
    const float radius = 0.5f * DIAM;
    if (radial_dist > radius) return;

    const idx_t idx3_in = gpu_idx_global3(x,y,z);
    d.uz[idx3_in] = U_JET;
    d.phi[idx3_in] = 1.0f;
}

__global__ void gpuInitDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;

    const idx_t idx3 = gpu_idx_global3(x,y,z);

    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        d.f[gpu_idx_global4(x,y,z,Q)] = gpu_compute_equilibria(d.rho[idx3],d.ux[idx3],d.uy[idx3],d.uz[idx3],Q);
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        d.g[gpu_idx_global4(x+CIX[Q],y+CIY[Q],z+CIZ[Q],Q)] = gpu_compute_truncated_equilibria(d.phi[idx3],d.ux[idx3],d.uy[idx3],d.uz[idx3],Q);
    }
} 

#ifdef DROPLET_CASE
__global__ void gpuInitDropletShape(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ || 
        x == 0 || x == NX-1 || 
        y == 0 || y == NY-1 || 
        z == 0 || z == NZ-1) return;
    const idx_t idx3 = gpu_idx_global3(x,y,z);
    
    const float center_x = (NX-1) * 0.5f;
    const float center_y = (NY-1) * 0.5f;
    const float center_z = (NZ-1) * 0.5f;

    const float dx = (x-center_x) / 2.0f, dy = y-center_y, dz = z-center_z;
    //const float dx= x-center_x, dy = y-center_y, dz = z-center_z;
    const float radial_dist = sqrtf(dx*dx + dy*dy + dz*dz);

    const float phi_val = 0.5f + 0.5f * tanhf(2.0f * (RADIUS-radial_dist) / 3.0f);
    d.phi[idx3] = phi_val;
}
#endif

