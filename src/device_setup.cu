#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitFieldsAndDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;
    const int idx3 = gpuIdxGlobal3(x,y,z);

    // no implicit initialization even though rho=1 and phi=0.
    // just going for safety here, as f and g could be simplified.
    d.rho[idx3] = 1.0f;
    d.phi[idx3] = 0.0f;
    float rho_val = d.rho[idx3];
    float phi_val = d.phi[idx3];
    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.f[idx4] = (W[Q] * rho_val) - W[Q];
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.g[idx4] = (W_G[Q] * phi_val) - W_G[Q];
    }
}

__constant__ float W[FLINKS];
__constant__ float W_G[GLINKS];

__constant__ int CIX[FLINKS], CIY[FLINKS], CIZ[FLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE =        NX * NY * NZ          * sizeof(float);            
    size_t F_DIST_SIZE = NX * NY * NZ * FLINKS * sizeof(float); 
    size_t G_DIST_SIZE = NX * NY * NZ * GLINKS * sizeof(float); 

    checkCudaErrors(hipMalloc(&lbm.phi,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ind,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.f,     F_DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g,     G_DIST_SIZE));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W),   &H_W,   FLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W_G), &H_W_G, GLINKS * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX),   &H_CIX,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY),   &H_CIY,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ),   &H_CIZ,   FLINKS * sizeof(int)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

