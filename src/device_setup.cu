#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitFieldsAndDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;
    const int idx3 = gpuIdxGlobal3(x,y,z);

    d.rho[idx3] = 1.0f;
    float rho_val = d.rho[idx3];
    float phi_val = d.phi[idx3];
    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.f[idx4] = to_dtype(W[Q] * rho_val - W[Q]);
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.g[idx4] = to_dtype(W_G[Q] * phi_val - W_G[Q]);
    }
}

__constant__ float W[FLINKS];
__constant__ float W_G[GLINKS];

__constant__ int CIX[FLINKS], CIY[FLINKS], CIZ[FLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE =        NX * NY * NZ          * sizeof(float);            
    size_t F_DIST_SIZE = NX * NY * NZ * FLINKS * sizeof(dtype); 
    size_t G_DIST_SIZE = NX * NY * NZ * GLINKS * sizeof(dtype); 

    checkCudaErrors(hipMalloc(&lbm.phi,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.f,     F_DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g,     G_DIST_SIZE));

    checkCudaErrors(hipMemset(lbm.phi,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ux,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uy,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uz,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.normx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normz, 0, SIZE));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W),   &H_W,   FLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W_G), &H_W_G, GLINKS * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX),   &H_CIX,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY),   &H_CIY,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ),   &H_CIZ,   FLINKS * sizeof(int)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

