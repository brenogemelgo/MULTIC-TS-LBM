#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void gpuInitFieldsAndDistributions(LBMFields d) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    const int z = threadIdx.z + blockIdx.z * blockDim.z;

    if (x >= NX || y >= NY || z >= NZ) return;
    const int idx3 = gpuIdxGlobal3(x,y,z);

    // no implicit initialization even though rho=1 and phi=0.
    // just going for safety here, as f and g could be simplified.
    d.rho[idx3] = 1.0f;
    float rho_val = d.rho[idx3];
    float phi_val = d.phi[idx3];
    #pragma unroll FLINKS
    for (int Q = 0; Q < FLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.f[idx4] = W[Q] * rho_val;
    }
    #pragma unroll GLINKS
    for (int Q = 0; Q < GLINKS; ++Q) {
        const int idx4 = gpuIdxGlobal4(x,y,z,Q);
        d.g[idx4] = W_G[Q] * phi_val;
    }
}


__constant__ float CSSQ;
__constant__ float OMEGA;
__constant__ float GAMMA;
__constant__ float INT_W;
__constant__ float SIGMA;
__constant__ float COEFF_HE;

__constant__ float W[FLINKS];
__constant__ float W_G[GLINKS];

__constant__ int CIX[FLINKS], CIY[FLINKS], CIZ[FLINKS];

#ifdef PERTURBATION
    __constant__ float DATAZ[200];
#endif

LBMFields lbm;
                                         
// =============================================================================================================================================================== //

void initDeviceVars() {
    size_t SIZE =        NX * NY * NZ          * sizeof(float);            
    size_t F_DIST_SIZE = NX * NY * NZ * FLINKS * sizeof(float); 
    size_t G_DIST_SIZE = NX * NY * NZ * GLINKS * sizeof(float); 

    checkCudaErrors(hipMalloc(&lbm.phi,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.rho,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ux,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uy,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.uz,    SIZE));
    checkCudaErrors(hipMalloc(&lbm.normx, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normy, SIZE));
    checkCudaErrors(hipMalloc(&lbm.normz, SIZE));
    checkCudaErrors(hipMalloc(&lbm.ind,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffx,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffy,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.ffz,   SIZE));
    checkCudaErrors(hipMalloc(&lbm.f,     F_DIST_SIZE));
    checkCudaErrors(hipMalloc(&lbm.g,     G_DIST_SIZE));

    checkCudaErrors(hipMemset(lbm.phi,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ux,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uy,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.uz,    0, SIZE));
    checkCudaErrors(hipMemset(lbm.normx, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normy, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.normz, 0, SIZE));
    checkCudaErrors(hipMemset(lbm.ind,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffx,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffy,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.ffz,   0, SIZE));
    checkCudaErrors(hipMemset(lbm.f,     0, F_DIST_SIZE));
    checkCudaErrors(hipMemset(lbm.g,     0, G_DIST_SIZE));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CSSQ),     &H_CSSQ,     sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(OMEGA),    &H_OMEGA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(INT_W),    &H_INT_W,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(GAMMA),    &H_GAMMA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(SIGMA),    &H_SIGMA,    sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(COEFF_HE), &H_COEFF_HE, sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W),   &H_W,   FLINKS * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(W_G), &H_W_G, GLINKS * sizeof(float)));

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIX),   &H_CIX,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIY),   &H_CIY,   FLINKS * sizeof(int)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(CIZ),   &H_CIZ,   FLINKS * sizeof(int)));

    #ifdef PERTURBATION
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(DATAZ), &H_DATAZ, 200 * sizeof(float)));
    #endif

    getLastCudaError("initDeviceVars: post-initialization");
}

