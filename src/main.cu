#include "kernels.cuh"
#include "host_functions.cuh"

int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Error: Usage: " << argv[0] << " <velocity set> <ID>" << std::endl;
        return 1;
    }
    std::string VELOCITY_SET = argv[1];
    std::string SIM_ID = argv[2];

    std::string SIM_DIR = createSimulationDirectory(VELOCITY_SET,SIM_ID);
    //computeAndPrintOccupancy();
    initDeviceVars();

    // ================================================================================================== //

    dim3 threadsPerBlock(BLOCK_SIZE_X,BLOCK_SIZE_Y,BLOCK_SIZE_Z);
    dim3 numBlocks((NX + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (NY + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   (NZ + threadsPerBlock.z - 1) / threadsPerBlock.z);

    dim3 threadsPerBlockZ(BLOCK_SIZE_X*2,BLOCK_SIZE_Y*2);  
    dim3 numBlocksZ((NX + threadsPerBlockZ.x - 1) / threadsPerBlockZ.x,
                    (NY + threadsPerBlockZ.y - 1) / threadsPerBlockZ.y);
                    
    hipStream_t mainStream;
    checkCudaErrors(hipStreamCreate(&mainStream));

    #ifdef DROPLET_CASE
        gpuInitDropletShape<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
        getLastCudaError("gpuInitDropletShape");
    #endif
    gpuInitFieldsAndDistributions<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
    getLastCudaError("gpuInitFieldsAndDistributions");

    auto START_TIME = std::chrono::high_resolution_clock::now();
    for (int STEP = 0; STEP <= NSTEPS ; ++STEP) {
        std::cout << "Passo " << STEP << " de " << NSTEPS << " iniciado..." << std::endl;

        // ========================= GRADIENTS & FORCES ========================= //

            gpuPhi<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
            getLastCudaError("gpuPhi");
            gpuGradients<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuGradients");
            gpuForces<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuForces");
        
        // ====================================================================== //
        
        // ========================= COLLISION & STREAMING ========================= //
        
            gpuCollisionStream<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuCollisionStream");
            gpuEvolvePhaseField<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
            getLastCudaError("gpuEvolvePhaseField");

        // ========================================================================= //    

        // =================================== BOUNDARIES =================================== //

            #ifdef JET_CASE
                gpuApplyInflow<<<numBlocksZ,threadsPerBlockZ,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm,STEP); 
                getLastCudaError("gpuApplyInflow");
                gpuApplyOutflow<<<numBlocksZ,threadsPerBlockZ,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
                getLastCudaError("gpuApplyOutflow");
                //gpuReconstructBoundaries<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
                //getLastCudaError("gpuReconstructBoundaries");
                //gpuApplyPeriodicXY<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm);
                //getLastCudaError("gpuApplyPeriodicXY");
            #elif defined(DROPLET_CASE)
                gpuReconstructBoundaries<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm); 
                getLastCudaError("gpuReconstructBoundaries");
            #endif

        // ================================================================================== //

        // =================================== DERIVED FIELDS =================================== //

            //gpuDerivedFields<<<numBlocks,threadsPerBlock,DYNAMIC_SHARED_SIZE,mainStream>>> (lbm,dfields); 
            //getLastCudaError("gpuDerivedFields");

        // ====================================================================================== //

        checkCudaErrors(hipDeviceSynchronize());

        if (STEP % MACRO_SAVE == 0) {

            copyAndSaveToBinary(lbm.phi,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"phi");
            copyAndSaveToBinary(lbm.uz,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"uz");
            //copyAndSaveToBinary(dfields.vorticity_mag,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"vorticity_mag");
            //copyAndSaveToBinary(dfields.q_criterion,NX*NY*NZ,SIM_DIR,SIM_ID,STEP,"q_criterion");

            std::cout << "Passo " << STEP << ": Dados salvos em " << SIM_DIR << std::endl;
        }
    }
    auto END_TIME = std::chrono::high_resolution_clock::now();

    checkCudaErrors(hipStreamDestroy(mainStream));

    // lbmfields
    hipFree(lbm.f); 
    hipFree(lbm.g);
    hipFree(lbm.phi); 
    hipFree(lbm.rho);
    hipFree(lbm.ind);
    hipFree(lbm.normx);
    hipFree(lbm.normy); 
    hipFree(lbm.normz);
    hipFree(lbm.ux); 
    hipFree(lbm.uy); 
    hipFree(lbm.uz);
    hipFree(lbm.pxx);
    hipFree(lbm.pyy);
    hipFree(lbm.pzz);
    hipFree(lbm.pxy);
    hipFree(lbm.pxz);
    hipFree(lbm.pyz);
    hipFree(lbm.ffx); 
    hipFree(lbm.ffy); 
    hipFree(lbm.ffz);

    // derivedfields
    hipFree(dfields.vorticity_mag);
    hipFree(dfields.q_criterion);

    std::chrono::duration<double> ELAPSED_TIME = END_TIME - START_TIME;
    long long TOTAL_CELLS = static_cast<long long>(NX) * NY * NZ * NSTEPS;
    double MLUPS = static_cast<double>(TOTAL_CELLS) / (ELAPSED_TIME.count() * 1e6);

    std::cout << "\n// =============================================== //\n";
    std::cout << "     Total execution time    : " << ELAPSED_TIME.count() << " seconds\n";
    std::cout << "     Performance             : " << MLUPS << " MLUPS\n";
    std::cout << "// =============================================== //\n" << std::endl;

    generateSimulationInfoFile(SIM_DIR,SIM_ID,VELOCITY_SET,NSTEPS,MACRO_SAVE,TAU,MLUPS);
    getLastCudaError("Final sync");
    return 0;
}
